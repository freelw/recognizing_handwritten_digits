#include <iostream>
#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <string.h>

#define TILE_WIDTH 32

__global__ void matrixmul(
    float *Md, float *Nd, float *Pd, int width) {
    //naive
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    for (int k = 0; k < width; ++k) {
        sum += Md[row * width + k] * Nd[k * width + col];
    }
    Pd[row * width + col] = sum;
}

void naive_matrixmul(float *Md, float *Nd, float *Pd, int width) {
    // Naive matrix multiplication
    for (int i = 0; i < width; ++i) {
        for (int j = 0; j < width; ++j) {
            Pd[i * width + j] = 0;
            for (int k = 0; k < width; ++k) {
                Pd[i * width + j] += Md[i * width + k] * Nd[k * width + j];
            }
        }
    }
}

bool check(float *h_output, float *res, int size) {
    for (int i = 0; i < size; ++i) {
        if (fabs(h_output[i] - res[i]) > 1e-3) {
            std::cout << "Error: " << "[" << i << "] " << h_output[i] << " != " << res[i] << std::endl;
            return false;
        }
    }
    return true;
}

int main() {
    const int WIDTH = 1024;
    float *h_Md = (float *)malloc(WIDTH*WIDTH*sizeof(float));
    float *h_Nd = (float *)malloc(WIDTH*WIDTH*sizeof(float));
    float *h_Pd = (float *)malloc(WIDTH*WIDTH*sizeof(float));
    float *h_res = (float *)malloc(WIDTH*WIDTH*sizeof(float));
    float *d_Md;
    float *d_Nd;
    float *d_Pd;

    hipMalloc((void **)&d_Md, WIDTH*WIDTH*sizeof(float));
    hipMalloc((void **)&d_Nd, WIDTH*WIDTH*sizeof(float));
    hipMalloc((void **)&d_Pd, WIDTH*WIDTH*sizeof(float));

    for (int i = 0; i < WIDTH*WIDTH; i++) {
        h_Md[i] = 2 * (float)drand48() - 1.0;
        h_Nd[i] = 2 * (float)drand48() - 1.0;
    }
    naive_matrixmul(h_Md, h_Nd, h_res, WIDTH);
    
    dim3 gridDim(
        (WIDTH + TILE_WIDTH - 1) / TILE_WIDTH,
        (WIDTH + TILE_WIDTH - 1) / TILE_WIDTH
    );
    dim3 blockDim(
        TILE_WIDTH,
        TILE_WIDTH
    );

    hipMemcpy(d_Md, h_Md, WIDTH*WIDTH*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_Nd, h_Nd, WIDTH*WIDTH*sizeof(float), hipMemcpyHostToDevice);
    matrixmul<<<gridDim, blockDim>>>(d_Md, d_Nd, d_Pd, WIDTH);
    hipMemcpy(h_Pd, d_Pd, WIDTH*WIDTH*sizeof(float), hipMemcpyDeviceToHost);

    if (check(h_Pd, h_res, WIDTH*WIDTH)) {
        std::cout << "Success!" << std::endl;
    } else {
        std::cout << "Failed!" << std::endl;
    }
    free(h_Md);
    free(h_Nd);
    free(h_Pd);
    hipFree(d_Md);
    hipFree(d_Nd);
    hipFree(d_Pd);
    return 0;
}