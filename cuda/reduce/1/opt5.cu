#include <iostream>
#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <string.h>

#define THREAD_PER_BLOCK 256

template <unsigned int blockSize>
__device__ void warpReduce(volatile float* cache, unsigned int tid){
    if (blockSize >= 64)cache[tid]+=cache[tid+32];
    if (blockSize >= 32)cache[tid]+=cache[tid+16];
    if (blockSize >= 16)cache[tid]+=cache[tid+8];
    if (blockSize >= 8)cache[tid]+=cache[tid+4];
    if (blockSize >= 4)cache[tid]+=cache[tid+2];
    if (blockSize >= 2)cache[tid]+=cache[tid+1];
}

template <unsigned int blockSize>
__global__ void reduce(float *d_in,float *d_out) {
    __shared__ float sdata[THREAD_PER_BLOCK];

    //each thread loads one element from global memory to shared mem
    unsigned int i=blockIdx.x*blockDim.x*2+threadIdx.x;
    unsigned int tid=threadIdx.x;
    sdata[tid]=d_in[i] + d_in[i+blockDim.x];
    __syncthreads();

    // do reduction in shared mem
    if (blockSize >= 512) {
        if (tid < 256) { 
            sdata[tid] += sdata[tid + 256]; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 256) {
        if (tid < 128) { 
            sdata[tid] += sdata[tid + 128]; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 128) {
        if (tid < 64) { 
            sdata[tid] += sdata[tid + 64]; 
        } 
        __syncthreads(); 
    }
    if (tid < 32) warpReduce<blockSize>(sdata, tid);

    // write result for this block to global mem
    if (tid == 0) d_out[blockIdx.x] = sdata[0];
}

bool check(float *h_output, float *res, int size) {
    for (int i = 0; i < size; ++i) {
        if (fabs(h_output[i] - res[i]) > 1e-3) {
            std::cout << "Error: " << "[" << i << "] " << h_output[i] << " != " << res[i] << std::endl;
            return false;
        }
    }
    return true;
}

int main() {
    const int N = 32 * 1024 * 1024;
    const int size = N * sizeof(float);
    int NUM_PER_BLOCK = 2*THREAD_PER_BLOCK;
    int block_num = N / NUM_PER_BLOCK;
    float *h_input = (float *)malloc(size);
    float *h_output = (float *)malloc(sizeof(float)*block_num);
    float *res = (float *)malloc(sizeof(float)*block_num);
    memset(res, 0, sizeof(float)*block_num);
    float *d_input;
    float *d_output;
    
    hipMalloc((void **)&d_input, size);
    hipMalloc((void **)&d_output, sizeof(float)*block_num);
    
    for (int i = 0; i < N; i++) {
        h_input[i] = 2 * (float)drand48() - 1.0;
        // h_input[i] = 1;
    }

    for (int i = 0; i < block_num; ++ i) {
        for (int j = 0; j < NUM_PER_BLOCK; ++ j) {
            res[i] += h_input[i * NUM_PER_BLOCK + j];
        }
    }
    
    dim3 gridDim(block_num);
    dim3 blockDim(THREAD_PER_BLOCK);

    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
    reduce<THREAD_PER_BLOCK><<<gridDim, blockDim>>>(d_input, d_output);
    hipMemcpy(h_output, d_output, sizeof(float)*block_num, hipMemcpyDeviceToHost);

    if (check(h_output, res, block_num)) {
        std::cout << "Success!" << std::endl;
    } else {
        std::cout << "Failed!" << std::endl;
    }
    free(h_input);
    free(h_output);
    free(res);
    hipFree(d_input);
    hipFree(d_output);
    return 0;
}