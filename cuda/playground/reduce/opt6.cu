#include <iostream>
#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <string.h>

#define THREAD_PER_BLOCK 256

template <unsigned int blockSize>
__device__ void warpReduce(volatile float* cache, unsigned int tid){
    if (blockSize >= 64)cache[tid]+=cache[tid+32];
    if (blockSize >= 32)cache[tid]+=cache[tid+16];
    if (blockSize >= 16)cache[tid]+=cache[tid+8];
    if (blockSize >= 8)cache[tid]+=cache[tid+4];
    if (blockSize >= 4)cache[tid]+=cache[tid+2];
    if (blockSize >= 2)cache[tid]+=cache[tid+1];
}

template <unsigned int blockSize, int NUM_PER_THREAD>
__global__ void reduce(float *d_in,float *d_out, unsigned int n){
    __shared__ float sdata[blockSize];

    // each thread loads NUM_PER_THREAD element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockSize * NUM_PER_THREAD) + threadIdx.x;

    sdata[tid] = 0;

    #pragma unroll
    for(int iter=0; iter<NUM_PER_THREAD; iter++){
        sdata[tid] += d_in[i+iter*blockSize];
    }
    
    __syncthreads();

    // do reduction in shared mem
    if (blockSize >= 512) {
        if (tid < 256) { 
            sdata[tid] += sdata[tid + 256]; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 256) {
        if (tid < 128) { 
            sdata[tid] += sdata[tid + 128]; 
        } 
        __syncthreads(); 
    }
    if (blockSize >= 128) {
        if (tid < 64) { 
            sdata[tid] += sdata[tid + 64]; 
        } 
        __syncthreads(); 
    }
    if (tid < 32) warpReduce<blockSize>(sdata, tid);
    
    // write result for this block to global mem
    if (tid == 0) d_out[blockIdx.x] = sdata[0];
}

bool check(float *h_output, float *res, int size) {
    for (int i = 0; i < size; ++i) {
        if (fabs(h_output[i] - res[i]) > 1e-2) {
            std::cout << "Error: " << "[" << i << "] " << h_output[i] << " != " << res[i] << std::endl;
            return false;
        }
    }
    return true;
}

int main() {
    const int N = 32 * 1024 * 1024;
    const int size = N * sizeof(float);
    
    const int block_num = 1024;
    const int NUM_PER_BLOCK = N / block_num;
    const int NUM_PER_THREAD = NUM_PER_BLOCK/THREAD_PER_BLOCK;
    float *h_input = (float *)malloc(size);
    float *h_output = (float *)malloc(sizeof(float)*block_num);
    float *res = (float *)malloc(sizeof(float)*block_num);
    memset(res, 0, sizeof(float)*block_num);
    float *d_input;
    float *d_output;
    
    hipMalloc((void **)&d_input, size);
    hipMalloc((void **)&d_output, sizeof(float)*block_num);
    
    for (int i = 0; i < N; i++) {
        h_input[i] = 2 * (float)drand48() - 1.0;
        // h_input[i] = 1;
    }

    for (int i = 0; i < block_num; ++ i) {
        for (int j = 0; j < NUM_PER_BLOCK; ++ j) {
            res[i] += h_input[i * NUM_PER_BLOCK + j];
        }
    }
    
    dim3 gridDim(block_num);
    dim3 blockDim(THREAD_PER_BLOCK);

    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
    reduce<THREAD_PER_BLOCK, NUM_PER_THREAD><<<gridDim, blockDim>>>(d_input, d_output, N);
    hipMemcpy(h_output, d_output, sizeof(float)*block_num, hipMemcpyDeviceToHost);

    if (check(h_output, res, block_num)) {
        std::cout << "Success!" << std::endl;
    } else {
        std::cout << "Failed!" << std::endl;
    }
    free(h_input);
    free(h_output);
    free(res);
    hipFree(d_input);
    hipFree(d_output);
    return 0;
}