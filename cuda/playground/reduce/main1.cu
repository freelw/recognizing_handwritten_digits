#include <iostream>
#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <string.h>

#define THREAD_PER_BLOCK 256
#define EPOCHS 8

__global__ void reduce(float *d_input, float *d_output) {
    float *sdata = d_input + blockIdx.x * blockDim.x;
    __shared__ float share[THREAD_PER_BLOCK];
    share[threadIdx.x] = sdata[threadIdx.x];
    __syncthreads();
    int pow = 1;
    for (int i = 0; i < EPOCHS; ++i) {
        pow *= 2;
        if (threadIdx.x % pow == 0) {
            share[threadIdx.x] += share[threadIdx.x + pow / 2];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        d_output[blockIdx.x] = share[0];
    }
}

bool check(float *h_output, float *res, int size) {
    for (int i = 0; i < size; ++i) {
        if (fabs(h_output[i] - res[i]) > 1e-3) {
            std::cout << "Error: " << "[" << i << "] " << h_output[i] << " != " << res[i] << std::endl;
            return false;
        }
    }
    return true;
}

int main() {
    const int N = 32 * 1024 * 1024;
    const int size = N * sizeof(float);
    int block_num = N / THREAD_PER_BLOCK;
    float *h_input = (float *)malloc(size);
    float *h_output = (float *)malloc(sizeof(float)*block_num);
    float *res = (float *)malloc(sizeof(float)*block_num);
    memset(res, 0, sizeof(float)*block_num);
    float *d_input;
    float *d_output;
    
    hipMalloc((void **)&d_input, size);
    hipMalloc((void **)&d_output, sizeof(float)*block_num);
    
    for (int i = 0; i < N; i++) {
        h_input[i] = 2 * (float)drand48() - 1.0;
        // h_input[i] = 1;
    }

    for (int i = 0; i < block_num; ++ i) {
        for (int j = 0; j < THREAD_PER_BLOCK; ++ j) {
            res[i] += h_input[i * THREAD_PER_BLOCK + j];
        }
    }
    dim3 gridDim(block_num);
    dim3 blockDim(THREAD_PER_BLOCK);

    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);
    reduce<<<gridDim, blockDim>>>(d_input, d_output);
    hipMemcpy(h_output, d_output, sizeof(float)*block_num, hipMemcpyDeviceToHost);

    if (check(h_output, res, block_num)) {
        std::cout << "Success!" << std::endl;
    } else {
        std::cout << "Failed!" << std::endl;
    }
    free(h_input);
    free(h_output);
    free(res);
    hipFree(d_input);
    hipFree(d_output);

    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}