#include "hip/hip_runtime.h"
#include "tensor.h"
#include "backends/cpu/cpu_ops.h"
#include "graph/node.h"
#include "common.h"
#include <iomanip>

const std::string RED = "\033[31m";
const std::string GREEN = "\033[32m";
const std::string RESET = "\033[0m";

void test_at() {
    init_backend();
    Tensor *input = allocTensor({2, 3}, "input");
    Tensor *w = allocTensor({3, 4}, "w");
    Tensor *wt = allocTensor({4, 3}, "wt");
    graph::Node *ni = graph::allocNode(input);
    graph::Node *nw = graph::allocNode(w);
    graph::Node *nwt = graph::allocNode(wt);
    auto res_wi = ni->at(nw);
    auto res_wti = ni->at(nwt->transpose());
    // printAllTensors();
    // printAllActions();
    allocMemAndInitTensors();
    input->fill(1.0f);
    for (int i = 0; i < 3; ++ i) {
        for (int j = 0; j < 4; ++ j) {
            float *loc_w = w->location({i, j});
            float *loc_wt = wt->location({j, i});
            float v = i * 4 + j;
            *loc_w = v;
            *loc_wt = v;
        }
    }
    gDoActions();
    auto res_wi_tensor = res_wi->get_tensor();
    auto res_wti_tensor = res_wti->get_tensor();
    auto res_wi_data = static_cast<float*>(res_wi_tensor->get_data());
    auto res_wti_data = static_cast<float*>(res_wti_tensor->get_data());
    const float eps = 1e-5f;
    bool succ = true;
    for (int i = 0; i < res_wi_tensor->length(); ++ i) {
        if (fabs(res_wi_data[i] - res_wti_data[i]) > eps) {
            succ = false;
            std::cerr << RED << "Error: res_wi[" << i << "] = " << res_wi_data[i]
                      << ", res_wti[" << i << "] = " << res_wti_data[i] << RESET << std::endl;
        }
    }
    if (succ) {
        std::cout << GREEN << "test_at succ " << RESET << std::endl;
    }
    sanitizeTensors();
    // // print res_wi shape
    // std::cout << "res_wi shape: ";
    // assert(res_wi_tensor->get_shape().size() == 2);
    // for (int i = 0; i < res_wi_tensor->get_shape().size(); ++ i) {
    //     std::cout << res_wi_tensor->get_shape()[i] << " ";
    // }
    // std::cout << std::endl;

    // // print res_wi data
    // std::cout << "res_wi data: " << std::endl;
    // for (int i = 0; i < res_wi_tensor->get_shape()[0]; ++ i) {
    //     for (int j = 0; j < res_wi_tensor->get_shape()[1]; ++ j) {
    //         std::cout << res_wi_data[i * res_wi_tensor->get_shape()[1] + j] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    freeAllActions();
    freeAllTensors();
    releaseTensorMem();
    release_backend();
}

void test_add() {
    init_backend();
    Tensor *input = allocTensor({3, 4}, "input");
    Tensor *w = allocTensor({3, 4}, "w");
    Tensor *wt = allocTensor({4, 3}, "wt");
    Tensor *res_wi_tensor = allocTensor({3, 4}, "res_wi");
    Tensor *res_wti_tensor = allocTensor({3, 4}, "res_wti");
    gCreateAction(
        new AddAction(input, w, res_wi_tensor)
    );
    gCreateAction(
        new AddAction(input, wt->transpose(), res_wti_tensor)
    );
    // printAllTensors();
    // printAllActions();
    allocMemAndInitTensors();
    input->fill(0.1f);
    for (int i = 0; i < 3; ++ i) {
        for (int j = 0; j < 4; ++ j) {
            float *loc_w = w->location({i, j});
            float *loc_wt = wt->location({j, i});
            float v = i * 4 + j;
            *loc_w = v;
            *loc_wt = v;
        }
    }
    gDoActions();
    auto res_wi_data = static_cast<float*>(res_wi_tensor->get_data());
    auto res_wti_data = static_cast<float*>(res_wti_tensor->get_data());
    const float eps = 1e-5f;
    bool succ = true;
    for (int i = 0; i < res_wi_tensor->length(); ++ i) {
        if (fabs(res_wi_data[i] - res_wti_data[i]) > eps) {
            succ = false;
            std::cerr << RED << "Error: res_wi[" << i << "] = " << res_wi_data[i]
                      << ", res_wti[" << i << "] = " << res_wti_data[i] << RESET << std::endl;
        }
    }
    if (succ) {
        std::cout << GREEN << "test_add succ" << RESET << std::endl;
    }

    sanitizeTensors();
    // // print res_wi shape
    // std::cout << "res_wi shape: " << res_wi_tensor->get_shape().size() << std::endl; 
    // assert(res_wi_tensor->get_shape().size() == 2);
    // for (int i = 0; i < res_wi_tensor->get_shape().size(); ++ i) {
    //     std::cout << res_wi_tensor->get_shape()[i] << " ";
    // }
    // std::cout << std::endl;

    // // print res_wi data
    // std::cout << "res_wi data: " << std::endl;
    // for (int i = 0; i < res_wi_tensor->get_shape()[0]; ++ i) {
    //     for (int j = 0; j < res_wi_tensor->get_shape()[1]; ++ j) {
    //         std::cout << res_wi_data[i * res_wi_tensor->get_shape()[1] + j] << " ";
    //     }
    //     std::cout << std::endl;
    // }

    freeAllActions();
    freeAllTensors();
    releaseTensorMem();
    release_backend();
}

void test_add_eq() {
    init_backend();
    Tensor *input = allocTensor({3, 4}, "input");
    Tensor *input1 = allocTensor({3, 4}, "input1");
    Tensor *w = allocTensor({3, 4}, "w");
    Tensor *wt = allocTensor({4, 3}, "wt");
    gCreateAction(
        new AddEqAction(input, w)
    );
    gCreateAction(
        new AddEqAction(input1, wt->transpose())
    );
    // printAllTensors();
    // printAllActions();
    allocMemAndInitTensors();
    input->fill(0.1f);
    input1->fill(0.1f);
    for (int i = 0; i < 3; ++ i) {
        for (int j = 0; j < 4; ++ j) {
            float *loc_w = w->location({i, j});
            float *loc_wt = wt->location({j, i});
            float v = i * 4 + j;
            *loc_w = v;
            *loc_wt = v;
        }
    }
    gDoActions();
    auto input_data = static_cast<float*>(input->get_data());
    auto input1_data = static_cast<float*>(input1->get_data());
    const float eps = 1e-5f;
    bool succ = true;
    for (int i = 0; i < input->length(); ++ i) {
        if (fabs(input_data[i] - input1_data[i]) > eps) {
            succ = false;
            std::cerr << RED << "Error: res_wi[" << i << "] = " << input_data[i]
                      << ", res_wti[" << i << "] = " << input1_data[i] << RESET << std::endl;
        }
    }
    if (succ) {
        std::cout << GREEN << "test_add_eq succ" << RESET << std::endl;
    }

    sanitizeTensors();

    freeAllActions();
    freeAllTensors();
    releaseTensorMem();
    release_backend();
}

void test_expand_add() {

    init_backend();
    Tensor *bias = allocTensor({4}, "bias");
    Tensor *w = allocTensor({3, 4}, "w");
    Tensor *wt = allocTensor({4, 3}, "wt");
    Tensor *res_wi_tensor = allocTensor({3, 4}, "res_wi");
    Tensor *res_wti_tensor = allocTensor({3, 4}, "res_wti");
    gCreateAction(
        new ExpandAddAction(w, bias, res_wi_tensor)
    );
    gCreateAction(
        new ExpandAddAction(wt->transpose(), bias, res_wti_tensor)
    );
    // printAllTensors();
    // printAllActions();
    allocMemAndInitTensors();
    bias->fill(0.1f);
    for (int i = 0; i < 3; ++ i) {
        for (int j = 0; j < 4; ++ j) {
            float *loc_w = w->location({i, j});
            float *loc_wt = wt->location({j, i});
            float v = i * 4 + j;
            *loc_w = v;
            *loc_wt = v;
        }
    }
    gDoActions();
    auto res_wi_data = static_cast<float*>(res_wi_tensor->get_data());
    auto res_wti_data = static_cast<float*>(res_wti_tensor->get_data());
    const float eps = 1e-5f;
    bool succ = true;
    for (int i = 0; i < res_wi_tensor->length(); ++ i) {
        if (fabs(res_wi_data[i] - res_wti_data[i]) > eps) {
            succ = false;
            std::cerr << RED << "Error: res_wi[" << i << "] = " << res_wi_data[i]
                      << ", res_wti[" << i << "] = " << res_wti_data[i] << RESET << std::endl;
        }
    }
    if (succ) {
        std::cout << GREEN << "test_expand_add succ" << RESET << std::endl;
    }

    sanitizeTensors();

    freeAllActions();
    freeAllTensors();
    releaseTensorMem();
    release_backend();
}

void test_mul() {
    init_backend();
    Tensor *input = allocTensor({3, 4}, "input");
    Tensor *w = allocTensor({3, 4}, "w");
    Tensor *wt = allocTensor({4, 3}, "wt");
    Tensor *res_wi_tensor = allocTensor({3, 4}, "res_wi");
    Tensor *res_wti_tensor = allocTensor({3, 4}, "res_wti");
    gCreateAction(
        new MulAction(input, w, res_wi_tensor)
    );
    gCreateAction(
        new MulAction(input, wt->transpose(), res_wti_tensor)
    );
    // printAllTensors();
    // printAllActions();
    allocMemAndInitTensors();
    input->fill(0.1f);
    for (int i = 0; i < 3; ++ i) {
        for (int j = 0; j < 4; ++ j) {
            float *loc_w = w->location({i, j});
            float *loc_wt = wt->location({j, i});
            float v = i * 4 + j;
            *loc_w = v;
            *loc_wt = v;
        }
    }
    gDoActions();
    auto res_wi_data = static_cast<float*>(res_wi_tensor->get_data());
    auto res_wti_data = static_cast<float*>(res_wti_tensor->get_data());
    const float eps = 1e-5f;
    bool succ = true;
    for (int i = 0; i < res_wi_tensor->length(); ++ i) {
        if (fabs(res_wi_data[i] - res_wti_data[i]) > eps) {
            succ = false;
            std::cerr << RED << "Error: res_wi[" << i << "] = " << res_wi_data[i]
                      << ", res_wti[" << i << "] = " << res_wti_data[i] << RESET << std::endl;
        }
    }
    if (succ) {
        std::cout << GREEN << "test_mul succ" << RESET << std::endl;
    }

    sanitizeTensors();

    freeAllActions();
    freeAllTensors();
    releaseTensorMem();
    release_backend();
}

void test_sum() {
    init_backend();
    Tensor *w = allocTensor({3, 4}, "w");
    Tensor *wt = allocTensor({4, 3}, "wt");
    Tensor *res_wi_tensor = allocTensor({3, 4}, "res_wi");
    Tensor *res_wti_tensor = allocTensor({3, 4}, "res_wti");
    gCreateAction(
        new SumAction(w, res_wi_tensor, 0)
    );
    gCreateAction(
        new SumAction(wt->transpose(), res_wti_tensor, 0)
    );
    // printAllTensors();
    // printAllActions();
    allocMemAndInitTensors();
    for (int i = 0; i < 3; ++ i) {
        for (int j = 0; j < 4; ++ j) {
            float *loc_w = w->location({i, j});
            float *loc_wt = wt->location({j, i});
            float v = i * 4 + j;
            *loc_w = v;
            *loc_wt = v;
        }
    }
    gDoActions();
    auto res_wi_data = static_cast<float*>(res_wi_tensor->get_data());
    auto res_wti_data = static_cast<float*>(res_wti_tensor->get_data());
    const float eps = 1e-5f;
    bool succ = true;
    for (int i = 0; i < res_wi_tensor->length(); ++ i) {
        if (fabs(res_wi_data[i] - res_wti_data[i]) > eps) {
            succ = false;
            std::cerr << RED << "Error: res_wi[" << i << "] = " << res_wi_data[i]
                      << ", res_wti[" << i << "] = " << res_wti_data[i] << RESET << std::endl;
        }
    }
    if (succ) {
        std::cout << GREEN << "test_sum succ" << RESET << std::endl;
    }

    sanitizeTensors();

    freeAllActions();
    freeAllTensors();
    releaseTensorMem();
    release_backend();
}

void test_cross_entropy() {
    init_backend();
    Tensor *labels = allocTensor({3}, "input", INT32);
    Tensor *w = allocTensor({3, 4}, "w");
    Tensor *wt = allocTensor({4, 3}, "wt");
    Tensor *res_wi_tensor = allocTensor({1}, "res_wi");
    Tensor *res_wti_tensor = allocTensor({1}, "res_wti");
    Tensor *maxs_wi = allocTensor({3}, "maxs_wi");
    Tensor *sums_wi = allocTensor({3}, "sums_wi");
    Tensor *maxs_wti = allocTensor({3}, "maxs_wti");
    Tensor *sums_wti = allocTensor({3}, "sums_wti");
    gCreateAction(
        new CrossEntropyAction(w, labels, maxs_wi, sums_wi, res_wi_tensor)
    );
    gCreateAction(
        new CrossEntropyAction(wt->transpose(), labels, maxs_wti, sums_wti, res_wti_tensor)
    );
    // printAllTensors();
    // printAllActions();
    allocMemAndInitTensors();
    for (int i = 0; i < 3; ++ i) {
        int32_t *loc_labels = reinterpret_cast<int32_t*>(labels->location({i}));
        *loc_labels = i;
    }
    for (int i = 0; i < 3; ++ i) {
        for (int j = 0; j < 4; ++ j) {
            float *loc_w = w->location({i, j});
            float *loc_wt = wt->location({j, i});
            float v = i * 4 + j;
            *loc_w = v;
            *loc_wt = v;
        }
    }
    
    gDoActions();
    auto res_wi_data = static_cast<float*>(res_wi_tensor->get_data());
    auto res_wti_data = static_cast<float*>(res_wti_tensor->get_data());
    const float eps = 1e-5f;
    bool succ = true;
    for (int i = 0; i < res_wi_tensor->length(); ++ i) {
        if (fabs(res_wi_data[i] - res_wti_data[i]) > eps) {
            succ = false;
            std::cerr << RED << "Error: res_wi[" << i << "] = " << res_wi_data[i]
                      << ", res_wti[" << i << "] = " << res_wti_data[i] << RESET << std::endl;
        }
    }
    if (succ) {
        std::cout << GREEN << "test_cross_entropy succ" << RESET << std::endl;
    }

    sanitizeTensors();

    freeAllActions();
    freeAllTensors();
    releaseTensorMem();
    release_backend();
}

void test_cross_entropy_backward() {
    init_backend();
    Tensor *labels = allocTensor({3}, "input", INT32);
    Tensor *w = allocTensor({3, 4}, "w");
    Tensor *wt = allocTensor({4, 3}, "wt");
    Tensor *res_wi_tensor = allocTensor({1}, "res_wi");
    Tensor *res_wti_tensor = allocTensor({1}, "res_wti");
    Tensor *maxs_wi = allocTensor({3}, "maxs_wi");
    Tensor *sums_wi = allocTensor({3}, "sums_wi");
    Tensor *maxs_wti = allocTensor({3}, "maxs_wti");
    Tensor *sums_wti = allocTensor({3}, "sums_wti");
    Tensor *grad_wi = allocTensor({3, 4}, "grad_wi");
    Tensor *grad_wti = allocTensor({3, 4}, "grad_wti");
    gCreateAction(
        new CrossEntropyAction(w, labels, maxs_wi, sums_wi, res_wi_tensor)
    );
    gCreateAction(
        new CrossEntropyAction(wt->transpose(), labels, maxs_wti, sums_wti, res_wti_tensor)
    );
    gCreateAction(
        new CrossEntropyBackwardAction(w, labels, maxs_wi, sums_wi, grad_wi)
    );
    gCreateAction(
        new CrossEntropyBackwardAction(wt->transpose(), labels, maxs_wti, sums_wti, grad_wti)
    );
    // printAllTensors();
    // printAllActions();
    allocMemAndInitTensors();
    for (int i = 0; i < 3; ++ i) {
        int32_t *loc_labels = reinterpret_cast<int32_t*>(labels->location({i}));
        *loc_labels = i;
    }
    for (int i = 0; i < 3; ++ i) {
        for (int j = 0; j < 4; ++ j) {
            float *loc_w = w->location({i, j});
            float *loc_wt = wt->location({j, i});
            float v = i * 4 + j;
            *loc_w = v;
            *loc_wt = v;
        }
    }
    
    gDoActions();

    auto grad_wi_data = static_cast<float*>(grad_wi->get_data());
    auto grad_wti_data = static_cast<float*>(grad_wti->get_data());

    const float eps = 1e-5f;
    bool succ = true;
    for (int i = 0; i < grad_wi->length(); ++ i) {
        if (fabs(grad_wi_data[i] - grad_wti_data[i]) > eps) {
            std::cerr << RED << "Error: grad_wi[" << i << "] = " << grad_wi_data[i]
                      << ", grad_wti[" << i << "] = " << grad_wti_data[i] << RESET << std::endl;
            succ = false;
        }
    }
    if (succ) {
        std::cout << GREEN << "test_cross_entropy_backward succ" << RESET << std::endl;
    }
    sanitizeTensors();
    freeAllActions();
    freeAllTensors();
    releaseTensorMem();
    release_backend();
}

void test_bp() {
    init_backend();
    Tensor *input = allocTensor({1, 2}, "input");
    Tensor *w = allocTensor({3, 2}, "w");
    Tensor *bias = allocTensor({3}, "bias");
    Tensor *w1 = allocTensor({3, 3}, "w1");
    Tensor *bias1 = allocTensor({3}, "bias1");

    graph::Node *ni = graph::allocNode(input);
    graph::Node *nw = graph::allocNode(w);
    graph::Node *nb = graph::allocNode(bias);
    graph::Node *nw1 = graph::allocNode(w1);
    graph::Node *nb1 = graph::allocNode(bias1);

    Tensor *labels = allocTensor({1}, "labels", INT32);
    auto foward_res0 = ni->at(nw->transpose())
        ->expand_add(nb)->relu();
    auto foward_res1 = foward_res0
        ->at(nw1->transpose())
        ->expand_add(nb1);
    auto nres = foward_res1
        ->CrossEntropy(labels);

    zero_grad();
    nres->backward();
    // printAllTensors();
    // printAllActions();
    allocMemAndInitTensors();

    float *input_data = static_cast<float*>(input->get_data());
    input_data[0] = 10.0f;
    input_data[1] = 11.0f;

    int32_t *labels_data = static_cast<int32_t*>(labels->get_data());
    labels_data[0] = 1;

    float *w_data = static_cast<float*>(w->get_data());
    for (int i = 0; i < w->length(); ++i) {
        w_data[i] = 0.1f;
    }

    float *bias_data = static_cast<float*>(bias->get_data());
    for (int i = 0; i < bias->length(); ++i) {
        bias_data[i] = 0.1f;
    }

    float *w1_data = static_cast<float*>(w1->get_data());
    for (int i = 0; i < w1->length(); ++i) {
        w1_data[i] = 0.1f;
    }

    float *bias1_data = static_cast<float*>(bias1->get_data());
    for (int i = 0; i < bias1->length(); ++i) {
        bias1_data[i] = 0.1f;
    }

    w_data[0] = 0.9f;
    w_data[1*w->get_shape()[1]] = -0.9f;

    w1_data[0] = 0.9f;
    w1_data[1*w1->get_shape()[1]] = -0.9f;

    gDoActions();

    // // print forward result
    // std::cout << "forward result0: " << std::endl;
    // for (int i = 0; i < foward_res0->get_tensor()->length(); ++i) {
    //     std::cout << static_cast<float*>(foward_res0->get_tensor()->get_data())[i] << " ";
    // }
    // std::cout << std::endl;
    // std::cout << "forward result1: " << std::endl;
    // for (int i = 0; i < foward_res1->get_tensor()->length(); ++i) {
    //     std::cout << static_cast<float*>(foward_res1->get_tensor()->get_data())[i] << " ";
    // }
    // std::cout << std::endl;

    // std::cout << "loss : " << std::setprecision(8) << static_cast<float*>(nres->get_tensor()->get_data())[0] << std::endl;
    const float eps = 1e-5f;
    bool loss_succ = fabs(static_cast<float*>(nres->get_tensor()->get_data())[0] - 18.360287f) < eps;
    if (loss_succ) {
        std::cout << GREEN << "test_cross_entropy succ" << RESET << std::endl;
    } else {
        std::cout << RED << "test_cross_entropy failed" << RESET << std::endl;
    }

    auto nw_grad = nw->get_grad();
    auto nb_grad = nb->get_grad();
    auto nw1_grad = nw1->get_grad();
    auto nb1_grad = nb1->get_grad();

    // print gradient
    bool nw_grad_succ = true;
    float nw_grad_ans[3][2] {
        17.997713,  19.797485,
        0.0000e+00,  0.0000e+00,
        -2.3890e-08, -2.6279e-08
    };
    for (int i = 0; i < nw_grad->get_shape()[0]; ++i) {
        for (int j = 0; j < nw_grad->get_shape()[1]; ++j) {
            float *loc_grad = static_cast<float*>(nw_grad->location({i, j}));
            if (fabs(*loc_grad - nw_grad_ans[i][j]) > eps) {
                std::cerr << std::setprecision(8) << RED << "Error: nw_grad[" << i << "][" << j << "] = " << *loc_grad
                          << ", nw_grad_ans[" << i << "][" << j << "] = " << nw_grad_ans[i][j] << RESET << std::endl;
                nw_grad_succ = false;
            }
        }
    }
    
    if (nw_grad_succ) {
        std::cout << GREEN << "test_cross_entropy nw_grad succ" << RESET << std::endl;
    }

    // print nb_grad

    // for (int i = 0; i < nb_grad->get_shape()[0]; ++i) {
    //     float *loc_grad = static_cast<float*>(nb_grad->location({i}));
    //     std::cout << std::setprecision(8) << "nb_grad[" << i << "] = " << *loc_grad << std::endl;
    // }
    bool nb_grad_succ = true;
    float nb_grad_ans[3] = {
        1.7997713,
        0.0000e+00,
        -2.3810571e-09
    };
    
    for (int i = 0; i < nb_grad->get_shape()[0]; ++i) {
        float *loc_grad = static_cast<float*>(nb_grad->location({i}));
        if (fabs(*loc_grad - nb_grad_ans[i]) > eps) {
            std::cerr << std::setprecision(8) << RED << "Error: nb_grad[" << i << "] = " << *loc_grad
                      << ", nb_grad_ans[" << i << "] = " << nb_grad_ans[i] << RESET << std::endl;
            nb_grad_succ = false;
        }
    }

    if (nb_grad_succ) {
        std::cout << GREEN << "test_cross_entropy nb_grad succ" << RESET << std::endl;
    }

    float nw1_grad_ans[3][3] = {
        10.197085, 0, 2.1993711,
        -10.200001, 0, -2.1999998,
        0.002914961, 0, 0.00062871695
    };

    bool nbw1_grad_succ = true;

    for (int i = 0; i < nw1_grad->get_shape()[0]; ++i) {
        for (int j = 0; j < nw1_grad->get_shape()[1]; ++j) {
            float *loc_grad = static_cast<float*>(nw1_grad->location({i, j}));
            if (fabs(*loc_grad - nw1_grad_ans[i][j]) > eps) {
                std::cerr << std::setprecision(8) << RED << "Error: nw1_grad[" << i << "][" << j << "] = " << *loc_grad
                          << ", nw1_grad_ans[" << i << "][" << j << "] = " << nw1_grad_ans[i][j] << RESET << std::endl;
                nbw1_grad_succ = false;
            }
        }
    }

    if (nbw1_grad_succ) {
        std::cout << GREEN << "test_cross_entropy nw1_grad succ" << RESET << std::endl;
    }

    float nb1_grad_ans[3] = {
        0.9997142,
        -1,
        0.00028578046
    };

    bool nb1_grad_succ = true;
    for (int i = 0; i < nb1_grad->get_shape()[0]; ++i) {
        float *loc_grad = static_cast<float*>(nb1_grad->location({i}));
        if (fabs(*loc_grad - nb1_grad_ans[i]) > eps) {
            std::cerr << std::setprecision(8) << RED << "Error: nb1_grad[" << i << "] = " << *loc_grad
                      << ", nb1_grad_ans[" << i << "] = " << nb1_grad_ans[i] << RESET << std::endl;
            nb1_grad_succ = false;
        }
    }

    if (nb1_grad_succ) {
        std::cout << GREEN << "test_cross_entropy nb1_grad succ" << RESET << std::endl;
    }

    sanitizeTensors();
    freeAllActions();
    freeAllTensors();
    releaseTensorMem();
    release_backend();
}

void test() {
    test_at();
    test_add();
    test_add_eq();
    test_expand_add();
    test_mul();
    test_sum();
    test_cross_entropy();
    test_cross_entropy_backward();
    test_bp();
}

int main() {
    test();
    freeAllTensors();
    freeAllTensorViews();
    graph::freeAllEdges();
    graph::freeAllNodes();
    freeAllActions();
    return 0;
}