#include "hip/hip_runtime.h"
#include "matrix.cuh"

#include <iostream>
#include <assert.h>
#include <string.h>
#include <vector>
#include <omp.h> // Include OpenMP header
#include <random>
#include <chrono>

Matrix::Matrix(Shape _shape)
        : initialized(false),
        allocated(false),
        shape(_shape) {
    data = new DATATYPE[shape.size()];
    allocated = true;
    zero();
}

Matrix::Matrix(const Matrix &m):
    initialized(m.initialized),
    allocated(false),
    shape(m.shape) {
    assert(initialized);
    data = new DATATYPE[shape.size()];
    allocated = true;
    memcpy(data, m.data, sizeof(DATATYPE) * shape.rowCnt * shape.colCnt);
}

Matrix::~Matrix() {
    assert(initialized && allocated);
    delete [] data;
    data = nullptr;
}

Matrix *Matrix::zero() {
    assert(allocated);
    memset(data, 0, sizeof(DATATYPE) * shape.size());
    initialized = true;
    return this;
}

bool Matrix::checkShape(const Matrix &m) {
    if (!(this->getShape() == m.getShape())) {
        std::cerr << 
            "matrix shape missmatch." << 
            this->getShape() << " vs " << m.getShape()<< 
            std::endl;
        assert(false);
    }
    if (!m.initialized) {
        std::cerr << "matrix not initialized..." << std::endl;
        assert(false);
    }
    return true;
}

ostream &operator<<(ostream &output, const Matrix &m) {
    if (!m.initialized) {
        output << "matrix not initialized." << endl;
        return output;
    }
    output << "[";
    for (uint i = 0; i < m.shape.rowCnt; ++ i) {
        if (i > 0) {
            output << " ";
        }
        output << "[";
        for (uint j = 0; j < m.shape.colCnt-1; ++ j) {
            output << m[i][j] << ", ";
        }
        output << m[i][m.shape.colCnt-1] << "]";
        if (i < m.shape.rowCnt-1) {
            output << endl;
        }
    }
    output << "]" << endl;
    return output;
}

Matrix *Matrix::expand_add(const Matrix &m) {
    assert(m.shape.rowCnt == shape.rowCnt);
    assert(m.shape.colCnt == 1);
    Matrix *res = allocTmpMatrix(this);
    for (uint i = 0; i < shape.rowCnt; ++i) {
        for (uint j = 0; j < shape.colCnt; ++j) {
            (*res)[i][j] += m[i][0];
        }
    }
    return res;
}

Matrix *Matrix::operator+(const Matrix &m) {
    checkShape(m);
    Matrix *res = allocTmpMatrix(this);
    #pragma omp parallel for num_threads(OMP_THREADS)
    for (uint i = 0; i < shape.rowCnt; ++i) {
        DATATYPE *m_data = m[i];
        DATATYPE *this_data = (*res)[i];
        for (uint j = 0; j < shape.colCnt; ++j) {
            this_data[j] += m_data[j];
        }
    }
    return res;
}

Matrix *Matrix::operator+=(const Matrix &m) {
    checkShape(m);
    DATATYPE *m_data = m.getData();
    DATATYPE *this_data = this->getData();

    const uint blockSize = 16; // Block size for cache optimization
    #pragma omp parallel for num_threads(OMP_THREADS)
    for (uint i = 0; i < shape.rowCnt; i += blockSize) {
        for (uint j = 0; j < shape.colCnt; j += blockSize) {
            for (uint ii = i; ii < std::min(i + blockSize, shape.rowCnt); ++ii) {
                for (uint jj = j; jj < std::min(j + blockSize, shape.colCnt); ++jj) {
                    this_data[ii * shape.colCnt + jj] += m_data[ii * shape.colCnt + jj];
                }
            }
        }
    }
    return this;
}

Matrix *Matrix::pow2() {
    Matrix *res = allocTmpMatrix(this);
    for (uint i = 0; i < shape.rowCnt; ++i) {
        for (uint j = 0; j < shape.colCnt; ++j) {
            auto &r = (*res)[i][j];
            r = std::pow(r, 2);
        }
    }
    return res;
}

Matrix *Matrix::operator+(DATATYPE dt) {
    Matrix *res = allocTmpMatrix(this);
    for (uint i = 0; i < shape.rowCnt; ++i) {
        for (uint j = 0; j < shape.colCnt; ++j) {
            (*res)[i][j] += dt;
        }
    }
    return res;
}

Matrix *Matrix::operator-(DATATYPE dt) {
    Matrix *res = allocTmpMatrix(this);
    for (uint i = 0; i < shape.rowCnt; ++i) {
        for (uint j = 0; j < shape.colCnt; ++j) {
            (*res)[i][j] -= dt;
        }
    }
    return res;
}

Matrix *Matrix::operator-() {
    Matrix *res = allocTmpMatrix(this);
    for (uint i = 0; i < shape.rowCnt; ++i) {
        for (uint j = 0; j < shape.colCnt; ++j) {
            auto &r = (*res)[i][j];
            r = -r;
        }
    }
    return res;
}

Matrix *operator-(DATATYPE v, const Matrix &m) {
    Matrix *res = allocTmpMatrix(m);
    for (uint i = 0; i < m.shape.rowCnt; ++i) {
        for (uint j = 0; j < m.shape.colCnt; ++j) {
            auto &r = (*res)[i][j];
            r = v-r;
        }
    }
    return res;
}

Matrix *Matrix::operator-(const Matrix &m) {
    checkShape(m);
    Matrix *res = allocTmpMatrix(this);
    for (uint i = 0; i < shape.rowCnt; ++i) {
        for (uint j = 0; j < shape.colCnt; ++j) {
            (*res)[i][j] -= m[i][j];
        }
    }
    return res;
}

Matrix *Matrix::operator-= (const Matrix &m) {
    checkShape(m);
    for (uint i = 0; i < shape.rowCnt; ++i) {
        for (uint j = 0; j < shape.colCnt; ++j) {
            (*this)[i][j] -= m[i][j];
        }
    }
    return this;
}

Matrix *Matrix::operator*(const Matrix &m) {
    checkShape(m);
    Matrix *res = allocTmpMatrix(this);
    for (uint i = 0; i < shape.rowCnt; ++i) {
        for (uint j = 0; j < shape.colCnt; ++j) {
            (*res)[i][j] *= m[i][j];
        }
    }
    return res;
}

Matrix *Matrix::operator*(DATATYPE v) {
    Matrix *res = allocTmpMatrix(this);
    for (uint i = 0; i < shape.rowCnt; ++i) {
        for (uint j = 0; j < shape.colCnt; ++j) {
            (*res)[i][j] *= v;
        }
    }
    return res;
}

Matrix *Matrix::operator*=(DATATYPE v) {
    for (uint i = 0; i < shape.rowCnt; ++i) {
        for (uint j = 0; j < shape.colCnt; ++j) {
            (*this)[i][j] *= v;
        }
    }
    return this;
}

Matrix *Matrix::operator/(DATATYPE v) {
    Matrix *res = allocTmpMatrix(this);
    for (uint i = 0; i < shape.rowCnt; ++i) {
        for (uint j = 0; j < shape.colCnt; ++j) {
            (*res)[i][j] /= v;
        }
    }
    return res;
}

Matrix *Matrix::Relu() {
    Matrix *res = allocTmpMatrix(this);
    #pragma omp parallel for num_threads(OMP_THREADS)
    for (uint i = 0; i < shape.rowCnt; ++i) {
        auto row = (*res)[i];
        for (uint j = 0; j < shape.colCnt; ++j) {
            row[j] = std::max(row[j], (DATATYPE)0);
        }
    }
    return res;
}

Matrix *Matrix::Relu_prime() {
    Matrix *res = allocTmpMatrix(this);
    #pragma omp parallel for num_threads(OMP_THREADS)
    for (uint i = 0; i < shape.rowCnt; ++i) {
        auto row = (*res)[i];
        for (uint j = 0; j < shape.colCnt; ++j) {
            row[j] = row[j] > 0 ? 1 : 0;
        }
    }
    return res;
}

Matrix *Matrix::tanh() {
    Matrix *res = allocTmpMatrix(this);
    #pragma omp parallel for num_threads(OMP_THREADS)
    for (uint i = 0; i < shape.rowCnt; ++i) {
        auto row = (*res)[i];
        for (uint j = 0; j < shape.colCnt; ++j) {
            row[j] = std::tanh(row[j]);
        }
    }
    return res;
}

Matrix *Matrix::tanh_prime() {
    return 1 - *(this->tanh()->pow2());
}

Matrix& Matrix::operator=(const Matrix &m) {
    assert(m.initialized);
    this->reShape(m.shape);
    for (uint i = 0; i < shape.rowCnt; ++i) {
        for (uint j = 0; j < shape.colCnt; ++j) {
            (*this)[i][j] = m[i][j];
        }
    }
    return *this;
}

DATATYPE *Matrix::operator[](unsigned int index) const {
    assert(index < shape.rowCnt);
    return (DATATYPE *)&(data[index*shape.colCnt]);
}

Shape Matrix::getShape() const {
    return shape;
}

Matrix *Matrix::at(const Matrix &m) {
    assert(m.shape.rowCnt == shape.colCnt);
    Matrix *res = allocTmpMatrix(Shape(shape.rowCnt, m.shape.colCnt));

    // DATATYPE *data = res->getData();
    // DATATYPE *m_data = m.getData();
    // DATATYPE *this_data = this->getData();

    // // use openmp for parallelization
    // #pragma omp parallel for collapse(2) num_threads(OMP_THREADS)
    // for (uint i = 0; i < shape.rowCnt; ++i) {
    //     for (uint j = 0; j < m.shape.colCnt; ++j) {
    //         DATATYPE sum = 0;
    //         for (uint k = 0; k < shape.colCnt; ++k) {
    //             sum += this_data[i * shape.colCnt + k] * m_data[k * m.shape.colCnt + j];
    //         }
    //         data[i * m.shape.colCnt + j] = sum;
    //     }
    // }

    DATATYPE *A = this->getData();
    DATATYPE *B = m.getData();
    DATATYPE *C = res->getData();

    #pragma omp parallel for num_threads(OMP_THREADS)
    for (uint i = 0; i < shape.rowCnt; ++i) {
        for (uint k = 0; k < shape.colCnt; ++k) {
            for (uint j = 0; j < m.shape.colCnt; ++j) {
                C[i * m.shape.colCnt + j] += A[i * shape.colCnt + k] * B[k * m.shape.colCnt + j];
            }
        }
    }

    return res;
}

Matrix *Matrix::transpose() {
    Matrix *res = allocTmpMatrix(Shape(shape.colCnt, shape.rowCnt));
    #pragma omp parallel for num_threads(OMP_THREADS)
    for (uint i = 0; i < shape.colCnt; ++ i) {
        DATATYPE *data = (*res)[i];
        for (uint j = 0; j < shape.rowCnt; ++ j) {
            data[j] = (*this)[j][i];
        }
    }
    return res;
}

bool Matrix::valid(uint x, uint y) const {
    return allocated && initialized && x < shape.rowCnt && y < shape.colCnt;
}

void Matrix::reShape(Shape _shape) {
    assert(allocated && initialized);
    delete []data;
    shape = _shape;
    data = new DATATYPE[shape.size()];
    zero();
}

Matrix *Matrix::assign(Matrix *other) {
    assert(allocated && initialized);
    checkShape(other->getShape());
    memcpy(data, other->data, sizeof(DATATYPE) * shape.size());
    return this;
}

Matrix *Matrix::sum(uint dim) {
    assert(dim == 1);
    if (dim == 1) {
        Matrix *res = allocTmpMatrix(Shape(shape.rowCnt, 1));
        for (uint i = 0; i < shape.rowCnt; ++ i) {
            for (uint j = 0; j < shape.colCnt; ++ j) {
                (*res)[i][0] += (*this)[i][j];
            }
        }
        return res;
    }
    return nullptr;
}

std::vector<Matrix *> Matrix::split(uint dim) {
    assert(dim == 1);
    if (dim == 1) {
        std::vector<Matrix *> res;
        for (uint i = 0; i < shape.colCnt; ++ i) {
            Matrix *m = allocTmpMatrix(Shape(shape.rowCnt, 1));
            for (uint j = 0; j < shape.rowCnt; ++ j) {
                (*m)[j][0] = (*this)[j][i];
            }
            res.push_back(m);
        }
        return res;
    }
    return {};
}

DATATYPE *Matrix::getData() const {
    return data;
}

Matrix *Matrix::fill(DATATYPE value) {
    for (uint i = 0; i < shape.rowCnt; ++ i) {
        for (uint j = 0; j < shape.colCnt; ++ j) {
            (*this)[i][j] = value;
        }
    }
    return this;
}

std::vector<uint> Matrix::argMax() {
    Shape shape = getShape();
    std::vector<uint> res;
    res.reserve(shape.colCnt);
    for (uint i = 0; i < shape.colCnt; ++ i) {
        uint max_index = 0;
        for (uint j = 1; j < shape.rowCnt; ++ j) {
            if ((*this)[j][i] > (*this)[max_index][i]) {
                max_index = j;
            }
        }
        res.push_back(max_index);
    }
    return res;
}

std::vector<DATATYPE> Matrix::avg() {
    Shape shape = getShape();
    std::vector<DATATYPE> res;
    res.reserve(shape.colCnt);
    for (uint i = 0; i < shape.colCnt; ++ i) {
        DATATYPE sum = 0;
        for (uint j = 0; j < shape.rowCnt; ++ j) {
            sum += (*this)[j][i];
        }
        res.push_back(sum/shape.rowCnt);
    }
    return res;
}

std::vector<DATATYPE> Matrix::var() {
    std::vector<DATATYPE> res;
    std::vector<DATATYPE> avg_res = this->avg();
    Shape shape = getShape();
    for (uint i = 0; i < shape.colCnt; ++ i) {
        DATATYPE sum = 0;
        auto avg_r = avg_res[i];
        for (uint j = 0; j < shape.rowCnt; ++ j) {
            sum += std::pow(((*this)[j][i] - avg_r), 2);
        }
        res.push_back(sum/shape.rowCnt);
    }
    return res;
}

DATATYPE _sigmoid(DATATYPE z) {
    return 1./(1.+exp(-z));
}

Matrix *Matrix::sigmoid() {
    Shape shape = getShape();
    Matrix *res = allocTmpMatrix(this);
    for (uint i = 0; i < shape.rowCnt; ++i) {
        for (uint j = 0; j < shape.colCnt; ++j) {
            (*res)[i][j] = _sigmoid((*res)[i][j]);
        }
    }
    return res;
}

Matrix *Matrix::sigmoid_prime() {
    return *sigmoid() * *(1 - *sigmoid());
}

std::vector<Matrix *> tmpMatrics;
Matrix *allocTmpMatrix(Matrix *m) {
    return allocTmpMatrix(*m);
}

Matrix *allocTmpMatrix(const Matrix &m) {
    Matrix *res = new Matrix(m);
    tmpMatrics.push_back(res);
    return res;
}

Matrix *allocTmpMatrix(const Shape & shape) {
    Matrix *res = new Matrix(shape);
    res->zero();
    tmpMatrics.push_back(res);
    return res;
}

void freeTmpMatrix() {
    for (auto p : tmpMatrics) {
        delete p;
    }
    tmpMatrics.clear();
}

void Matrix::init_weight(DATATYPE sigma, DATATYPE mean) {
    unsigned seed1 = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator_w(seed1);
    std::normal_distribution<DATATYPE> distribution_w(0.0, sigma);
    auto shape = getShape();
    for (uint i = 0; i < shape.rowCnt; ++ i) {
        for (uint j = 0; j < shape.colCnt; ++ j) {
            (*this)[i][j] = distribution_w(generator_w) + mean;
        }
    }
}

void Matrix::init_weight_uniform(DATATYPE sigma) {
    unsigned seed1 = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator_w(seed1);
    std::uniform_real_distribution<DATATYPE> distribution_w(-sigma, sigma);
    auto shape = getShape();
    for (uint i = 0; i < shape.rowCnt; ++ i) {
        for (uint j = 0; j < shape.colCnt; ++ j) {
            (*this)[i][j] = distribution_w(generator_w);
        }
    }
}


TrainingData::TrainingData(int input_layer_size, int _y)
    : y(_y) {  
    x = new Matrix(Shape(input_layer_size, 1));
    x->zero();
}

TrainingData::~TrainingData() {
    delete x;
}