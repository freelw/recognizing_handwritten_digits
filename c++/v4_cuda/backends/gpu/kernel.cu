#include "hip/hip_runtime.h"
#include "kernel.cuh"

__global__ void matrixmul(
    float *Md, float *Nd, float *Pd, int M, int N, int P) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float s_Md[TILE_WIDTH][TILE_WIDTH];
    __shared__ float s_Nd[TILE_WIDTH][TILE_WIDTH];
    
    for (int m = 0; m < (N + TILE_WIDTH - 1)/ TILE_WIDTH; ++m) {
        // Load data into shared memory
        s_Md[threadIdx.y][threadIdx.x] = row < M && m * TILE_WIDTH + threadIdx.x < N ? Md[row * N + m * TILE_WIDTH + threadIdx.x] : 0.f;
        s_Nd[threadIdx.y][threadIdx.x] = col < P && m * TILE_WIDTH + threadIdx.y < N ? Nd[(m * TILE_WIDTH + threadIdx.y) * P + col] : 0.f;
        __syncthreads();
        if (row >= M || col >= P) {
            
        } else {
            float sum = 0;
            for (int k = 0; k < TILE_WIDTH; ++k) {
                sum += s_Md[threadIdx.y][k] * s_Nd[k][threadIdx.x];
            }
            Pd[row * P + col] += sum;
        }
        __syncthreads();
    }
}

__global__ void expand_add_kernel(
    float *Md, float *Nd, int M, int N) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= M || col >= N) {
            
    } else {
       Md[row * N + col] += Nd[row];
    }
}

__global__ void relu_kernel(float *Md, int M) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= M) {
            
    } else {
        Md[index] = fmaxf(0, Md[index]);
    }
}

__global__ void add_eq_kernel(float *Md, float *Nd, int M, int N) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= M || col >= N) {
            
    } else {
       Md[row * N + col] += Nd[row * N + col];
    }
}