#include "cuda_ops.cuh"
#include "kernel.cuh"


bool GPUBackendOps::is_gpu() {
    return true;
}

void GPUBackendOps::cp_to_device(void* dst, const void* src, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}

void GPUBackendOps::cp_from_device(void* dst, const void* src, size_t size) {
    hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}

Matrix *GPUBackendOps::CrossEntropyLoss(
    Matrix *input,
    const std::vector<uint> &labels,
    std::vector<autograd_cuda::CrosEntropyInfo> &info) {
    std::cerr << "CrossEntropyLoss unimplemented" << std::endl;
    assert(false);
    return nullptr;
}

Matrix *GPUBackendOps::CrossEntropyLossMask(
    Matrix *input,
    const std::vector<uint> &labels,
    std::vector<autograd_cuda::CrosEntropyInfo> &info,
    const std::vector<bool> &mask) {
    std::cerr << "CrossEntropyLossMask unimplemented" << std::endl;
    assert(false);
    return nullptr;
}

Matrix *GPUBackendOps::Norm(
    Matrix *w,
    const std::vector<DATATYPE> &avg_res,
    const std::vector<DATATYPE> &var_res,
    DATATYPE eps) {
    std::cerr << "Norm unimplemented" << std::endl;
    assert(false);
    return nullptr;
}

Matrix *GPUBackendOps::Softmax(Matrix *w) {
    std::cerr << "Softmax unimplemented" << std::endl;
    assert(false);
    return nullptr;
}

std::vector<Matrix*> GPUBackendOps::split0(Matrix *w) {
    std::cerr << "split0 unimplemented" << std::endl;
    assert(false);
    return std::vector<Matrix*>();
}

std::vector<Matrix*> GPUBackendOps::split1(Matrix *w, uint step) {
    std::cerr << "split1 unimplemented" << std::endl;
    assert(false);
    return std::vector<Matrix*>();
}

void GPUBackendOps::CrossEntropyEdgeBackward(
    Matrix *w,
    Matrix *grad,
    const std::vector<uint> &labels,
    const std::vector<autograd_cuda::CrosEntropyInfo> &info) {
    std::cerr << "CrossEntropyEdgeBackward unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::CrossEntropyMaskEdgeBackward(
    Matrix *w,
    Matrix *grad,
    const std::vector<uint> &labels,
    const std::vector<autograd_cuda::CrosEntropyInfo> &info,
    const std::vector<bool> &mask) {
    std::cerr << "CrossEntropyMaskEdgeBackward unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::NormEdgeBackward(
    Matrix *w,
    Matrix *grad,
    const std::vector<DATATYPE> &avg_res,
    const std::vector<DATATYPE> &var_res,
    DATATYPE eps) {
    std::cerr << "NormEdgeBackward unimplemented" << std::endl;
    assert(false);
}

DATATYPE *GPUBackendOps::allocDeviceMem(size_t size) {
    DATATYPE *ret = nullptr;
    hipMalloc((void **)&ret, size);
    return ret;
}

void GPUBackendOps::releaseDeviceMem(DATATYPE *ptr) {
    assert(ptr != nullptr);
    hipFree(ptr);
}

void GPUBackendOps::expand_add(Matrix *w, const Matrix &m) {
    std::cerr << "expand_add unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_add(Matrix *w, const Matrix &m) {
    std::cerr << "operator_add unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::pow2(Matrix *w) {
    std::cerr << "pow2 unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_add_val(Matrix *w, DATATYPE v) {
    std::cerr << "operator_add_val unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_minus_val(Matrix *w, DATATYPE v) {
    std::cerr << "operator_minus_val unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_negative(Matrix *w) {
    std::cerr << "operator_negative unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_val_minus(DATATYPE v, Matrix *w) {
    std::cerr << "operator_val_minus unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_minus(Matrix *w, const Matrix &m) {
    std::cerr << "operator_minus unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_multiply(Matrix *w, const Matrix &m) {
    std::cerr << "operator_multiply unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_multiply_val(Matrix *w, DATATYPE v) {
    std::cerr << "operator_multiply_val unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_divide(Matrix *w, const Matrix &m) {
    std::cerr << "operator_divide unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_divide_val(Matrix *w, DATATYPE v) {
    std::cerr << "operator_divide_val unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_relu(Matrix *w) {
    std::cerr << "operator_relu unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_relu_prime(Matrix *w) {
    std::cerr << "operator_relu_prime unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_tanh(Matrix *w) {
    std::cerr << "operator_tanh unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_tanh_prime(Matrix *w) {
    std::cerr << "operator_tanh_prime unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_equal(Matrix *w, const Matrix &m) {
    std::cerr << "operator_equal unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_at(Matrix *res, Matrix *w, Matrix &m) {
    w->sync();
    m.sync();

    auto wshape = w->getShape();
    auto mshape = m.getShape();
    auto rshape = res->getShape();

    assert(wshape.colCnt == mshape.rowCnt);
    assert(rshape.rowCnt == wshape.rowCnt);

    const int M = wshape.rowCnt;
    const int N = wshape.colCnt;
    const int P = mshape.colCnt;
    
    dim3 gridDim(
        (P + TILE_WIDTH - 1) / TILE_WIDTH,
        (M + TILE_WIDTH - 1) / TILE_WIDTH
    );
    dim3 blockDim(
        TILE_WIDTH,
        TILE_WIDTH
    );
    DATATYPE *d_Md = w->getLowLevelDataDevice();
    DATATYPE *d_Nd = m.getLowLevelDataDevice();
    DATATYPE *d_Pd = res->getLowLevelDataDevice();

    matrixmul<<<gridDim, blockDim>>>(d_Md, d_Nd, d_Pd, M, N, P);
    res->increase_gpu_ver();
}

void GPUBackendOps::operator_transpose(Matrix *res, Matrix *w) {
    std::cerr << "operator_transpose unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_assign(Matrix *res, Matrix *w) {
    std::cerr << "operator_assign unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_sum(Matrix *res, Matrix *w) {
    std::cerr << "operator_sum unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_split(std::vector<Matrix *> &res, Matrix *w) {
    std::cerr << "operator_split unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_fill(Matrix *w, DATATYPE value) {
    std::cerr << "operator_fill unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_argMax(std::vector<uint> &res, Matrix *w) {
    std::cerr << "operator_argMax unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_avg(std::vector<DATATYPE> &res, Matrix *w) {
    std::cerr << "operator_avg unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_var(std::vector<DATATYPE> &res, Matrix *w) {
    std::cerr << "operator_var unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_sigmoid(Matrix *w) {
    std::cerr << "operator_sigmoid unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_sigmoid_prime(Matrix *w) {
    std::cerr << "operator_sigmoid_prime unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_init_weight(Matrix *w, DATATYPE sigma, DATATYPE mean) {
    std::cerr << "operator_init_weight unimplemented" << std::endl;
    assert(false);
}

void GPUBackendOps::operator_init_weight_uniform(Matrix *w, DATATYPE sigma) {
    std::cerr << "operator_init_weight_uniform unimplemented" << std::endl;
    assert(false);
}
